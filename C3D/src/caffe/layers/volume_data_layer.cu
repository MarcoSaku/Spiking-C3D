/*
 *
 *  Copyright (c) 2015, Facebook, Inc. All rights reserved.
 *
 *  Licensed under the Creative Commons Attribution-NonCommercial 3.0
 *  License (the "License"). You may obtain a copy of the License at
 *  https://creativecommons.org/licenses/by-nc/3.0/.
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  License for the specific language governing permissions and limitations
 *  under the License.
 *
 *
 */


#include <stdint.h>
#include <leveldb/db.h>
#include <pthread.h>

#include <string>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/volume_data_layer.hpp"

using std::string;

namespace c3d_caffe {

template <typename Dtype>
Dtype VolumeDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  // First, join the thread
  JoinPrefetchThread();
  // Copy the data
  CUDA_CHECK(hipMemcpy((*top)[0]->mutable_gpu_data(),
      prefetch_data_->cpu_data(), sizeof(Dtype) * prefetch_data_->count(),
      hipMemcpyHostToDevice));
  if (output_labels_) {
    CUDA_CHECK(hipMemcpy((*top)[1]->mutable_gpu_data(),
        prefetch_label_->cpu_data(), sizeof(Dtype) * prefetch_label_->count(),
        hipMemcpyHostToDevice));
  }
  // Start a new prefetch thread
  CreatePrefetchThread();
  return Dtype(0.);
}

INSTANTIATE_CLASS(VolumeDataLayer);

}  // namespace c3d_caffe