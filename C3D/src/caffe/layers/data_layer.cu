// Copyright 2014 BVLC and contributors.

#include <stdint.h>
#include <leveldb/db.h>
#include <pthread.h>

#include <string>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/vision_layers.hpp"

using std::string;

namespace c3d_caffe {

template <typename Dtype>
Dtype DataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  // First, join the thread
  JoinPrefetchThread();
  // Copy the data
  CUDA_CHECK(hipMemcpy((*top)[0]->mutable_gpu_data(),
      prefetch_data_->cpu_data(), sizeof(Dtype) * prefetch_data_->count(),
      hipMemcpyHostToDevice));
  if (output_labels_) {
    CUDA_CHECK(hipMemcpy((*top)[1]->mutable_gpu_data(),
        prefetch_label_->cpu_data(), sizeof(Dtype) * prefetch_label_->count(),
        hipMemcpyHostToDevice));
  }
  // Start a new prefetch thread
  CreatePrefetchThread();
  return Dtype(0.);
}

INSTANTIATE_CLASS(DataLayer);

}  // namespace c3d_caffe
